#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void testGPU(){
    printf("Hello this is GPU\n");
}

int main(){
    printf("Hi from CPU\n");
    
    testGPU<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}