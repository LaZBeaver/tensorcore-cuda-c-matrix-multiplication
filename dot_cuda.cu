#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void dot(float *a, float *b, float *c, int a_r, int b_c, int a_b_s){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < a_r && col < b_c){
        float tmp_sum = 0;
        for (int i = 0; i < a_b_s; i++){
            // a_c == b_r == a_b_s
            tmp_sum += a[row * a_b_s + i] * b[i * b_c + col];
        }
        c[row * b_c + col] = tmp_sum;
    }

}

void init(float *m, int r, int c){
    for (int i = 0; i < r * c; i++)
        m[i] = (float)rand() / RAND_MAX;
        // m[i] = rand() % 10;
        
}

void print(float *m, int r, int c){
    for (int i = 0; i < r; i++){
        for (int j = 0; j < c; j++)
            printf("%f ", m[i * c + j]);
        printf("\n");
    }
}

int main(){
    int a_r = 1024;
    int a_b_s = 1024;
    int b_c = 1024;
    
    size_t a_bytes = a_r * a_b_s * sizeof(float);
    size_t b_bytes = a_b_s * b_c * sizeof(float);
    size_t c_bytes = a_r * b_c * sizeof(float);

    float *a, *b, *c;

    hipMallocManaged(&a, a_bytes);
    hipMallocManaged(&b, b_bytes);
    hipMallocManaged(&c, c_bytes);

    init(a, a_r, a_b_s);
    init(b, a_b_s, b_c);
    
    int threads_per_dim_in_block = 16;
    
    int blocks_x = (b_c - 1) / threads_per_dim_in_block + 1;
    int blocks_y = (a_r - 1) / threads_per_dim_in_block + 1;
    dim3 BLOCKS(blocks_x, blocks_y, 1);
    
    dim3 THREADS(threads_per_dim_in_block, threads_per_dim_in_block, 1);
    

    dot<<<BLOCKS, THREADS>>>(a, b, c, a_r, b_c, a_b_s);

    hipDeviceSynchronize();

    // print(a, a_r, a_b_s);
    // print(b, a_b_s, b_c);
    // print(c, a_r, b_c);

    return 0;
}