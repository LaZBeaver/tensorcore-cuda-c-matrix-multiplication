#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void dot(float *a, float *b, float *c, int a_r, int b_c, int a_b_s){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < a_r && col < b_c){
        float tmp_sum = 0;
        for (int i = 0; i < a_b_s; i++){
            // a_c == b_r == a_b_s
            tmp_sum += a[row * a_b_s + i] * b[i * b_c + col];
        }
        c[row * b_c + col] = tmp_sum;
    }

}

__global__ void sum(float *a, float *b, float *c, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
        c[tid] = a[tid] + b[tid];
}

void init(float *m, int r, int c){
    for (int i = 0; i < r * c; i++)
        // m[i] = rand() % 10;
        m[i] = (float)rand() / RAND_MAX;
}

void print(float *m, int r, int c){
    for (int i = 0; i < r; i++){
        for (int j = 0; j < c; j++)
            printf("%f ", m[i * c + j]);
        printf("\n");
    }
}

int main(){
    int a_r = 1024;
    int a_b_s = 1024;
    int b_c = 1024;

    float *h_a;
    float *h_b;

    float *d_a;
    float *d_b;
    float *d_c;

    size_t a_bytes = a_r * 1 * sizeof(float);
    size_t b_bytes = 1 * b_c * sizeof(float);
    size_t c_bytes = a_r * b_c * sizeof(float);

    // A place to store all the partials to add them later all together
    float *partials[a_b_s];

    for (int i = 0; i < a_b_s; i++){
        
        h_a = (float*)malloc(a_bytes);
        h_b = (float*)malloc(b_bytes);
        partials[i] = (float*)malloc(c_bytes);

        init (h_a, a_r, 1);
        init (h_b, 1, b_c);

        hipMalloc(&d_a, a_bytes);
        hipMalloc(&d_b, b_bytes);
        hipMalloc(&d_c, c_bytes);

        hipMemcpy(d_a, h_a, a_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, b_bytes, hipMemcpyHostToDevice);

        int threads_per_dim_in_block = 16;

        int blocks_x = (b_c - 1) / threads_per_dim_in_block + 1;
        int blocks_y = (a_r - 1) / threads_per_dim_in_block + 1;

        dim3 BLOCKS(blocks_x, blocks_y);
        dim3 THREADS(threads_per_dim_in_block, threads_per_dim_in_block);

        dot<<<BLOCKS, THREADS>>>(d_a, d_b, d_c, a_r, b_c, 1);

        hipMemcpy(partials[i], d_c, c_bytes, hipMemcpyDeviceToHost);

        // printf("For %d col of a and row of b we have:\n", i);
        // print(h_a, a_r, 1);
        // print(h_b, 1, b_c);
        // print(partials[i], a_r, b_c);
        // printf("\n");

        free(h_a);
        free(h_b);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

    }

    hipDeviceSynchronize();

    // Now we have all the partials in the partials[] and we only need to sum the co-responding elements
    // for this we use the kernel that sums up to vectores and since the placment in menmory is actually
    // row majored we make a final result which is the same size as it needs to be then initialize it to
    // zero and then pass that with each elemt of partials[] to get the sum.

    float *result;
    result = (float*)malloc(c_bytes);
    for (int i = 0; i < a_r * b_c; i++)
        result[i] = 0;
    
    for (int i = 0; i < a_b_s; i++){

        hipMalloc(&d_a, c_bytes);
        hipMalloc(&d_b, c_bytes);
        hipMalloc(&d_c, c_bytes);

        hipMemcpy(d_a, partials[i], c_bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, result, c_bytes, hipMemcpyHostToDevice);

        int THREADS = 256;
        int BLOCKS = (a_r * b_c - 1) / THREADS + 1;

        sum<<<BLOCKS, THREADS>>>(d_a, d_b, d_c, a_r * b_c);

        hipMemcpy(result, d_c, c_bytes, hipMemcpyDeviceToHost);

        free(partials[i]);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);        
    
    }

    hipDeviceSynchronize();
    
    // print(result, a_r, b_c);
    
    return 0;
}