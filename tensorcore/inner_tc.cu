#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>

#define WARP_SIZE 32

// Matrix A = (M_tiles * K_tiles)
// Matrix B = (K_tiles * N_tiles)
// Matrix C & D = (M_tiles * N_tiles)
// Each tile is equal to 16 element

#define M 16
#define N 16
#define K 16

#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

#define M_TOTAL (M * M_TILES)
#define N_TOTAL (N * N_TILES)
#define K_TOTAL (K * K_TILES)

using namespace nvcuda;

void InitMatrix(half *A, half *B, float *C)
{
	for (int i = 0; i < M_TOTAL * K_TOTAL; i++)
		A[i] = __float2half((float)rand() / RAND_MAX);

	for (int i = 0; i < K_TOTAL * N_TOTAL; i++)
		B[i] = __float2half((float)rand() / RAND_MAX);

	for (int i = 0; i < M_TOTAL * N_TOTAL; i++)
		C[i] = 0.0f;
}



__global__ void WMMAF16TensorCore(half *A, half *B, float *C, float *D)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x)/WARP_SIZE;
	int iy = (blockIdx.y * blockDim.y + threadIdx.y);
	
	wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
	wmma::fragment<wmma::accumulator, M, N, K, float> ab_frag;
	wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;
	
	wmma::fill_fragment(ab_frag, 0.0f);

	// AB = A*B
	int a_col, a_row, b_col, b_row, c_col, c_row;
	a_row = ix * M;
	b_row = iy * N;
	for (int k=0; k<K_TOTAL; k+=K) {
		a_col = b_col = k;

		if (a_row < M_TOTAL && a_col < K_TOTAL && b_row < K_TOTAL && b_col < N_TOTAL) {
			// Load the inputs
			wmma::load_matrix_sync(a_frag, A + a_col + a_row * M_TOTAL, M_TOTAL);
			wmma::load_matrix_sync(b_frag, B + b_col + b_col * K_TOTAL, K_TOTAL);

			// Perform the matrix multiplication
			wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
		}
	}

	// D = AB + C
	c_col = b_row;
	c_row = a_row;
	if (c_row < M_TOTAL && c_col < N_TOTAL) {
		wmma::load_matrix_sync(c_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);

		for (int i = 0; i < c_frag.num_elements; i++) {
			c_frag.x[i] = ab_frag.x[i] + c_frag.x[i];
		}

		// Store the output
		wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag, N_TOTAL, wmma::mem_row_major);
	}
}


int main()
{

	half *A;
	half *B;
	float *C;
	float *D;

	hipMallocManaged((void **)&A, sizeof(half) * M_TOTAL * K_TOTAL);
	hipMallocManaged((void **)&B, sizeof(half) * K_TOTAL * N_TOTAL);
	hipMallocManaged((void **)&C, sizeof(float) * M_TOTAL * N_TOTAL);
	hipMallocManaged((void **)&D, sizeof(float) * M_TOTAL * N_TOTAL);
	
	InitMatrix(A, B, C);

	printf("A: %d x %d\n", M_TOTAL, K_TOTAL);
	printf("B: %d x %d\n", K_TOTAL, N_TOTAL);
	printf("C: %d x %d\n", M_TOTAL, N_TOTAL);
	
	dim3 gridDim, blockDim;
	blockDim.x = 4 * WARP_SIZE; 
	blockDim.y = 4;

	gridDim.x = (M_TOTAL + (M * blockDim.x / WARP_SIZE - 1)) / (M * blockDim.x / WARP_SIZE);
	gridDim.y = (N_TOTAL + N * blockDim.y - 1) / (N * blockDim.y);

	WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D);
	
	hipDeviceSynchronize();

	hipFree(A);
	hipFree(B);
	hipFree(C);
	hipFree(D);

	return 0;
}